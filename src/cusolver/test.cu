#include <stdio.h> 
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipsolver.h"
#include <iostream>

__device__ __managed__
  double A[] = {0.8147,    0.6324,    0.9575,    0.9572,
                0.9058,    0.0975,    0.9649,    0.4854,
                0.1270,    0.2785,    0.1576,    0.8003,
                0.9134,    0.5469,    0.9706,    0.1419};
__device__ __managed__
  double A2[] = {0.8147,    0.9058,    0.1270,    0.9134,
                0.6324,    0.0975,    0.2785,    0.5469,
                0.9575,    0.9649,    0.1576,    0.9706,
                0.9572,    0.4854,    0.8003,    0.1419};
  __device__ __managed__
  double b[] = {0.4218,
                0.9157,
                0.7922,
                0.9595};
  __device__ __managed__
  double x[] = { 17.2819,
                  0.8395,
                -15.9067,
                  1.0883};
  __device__ __managed__ double *Workspace;
  __device__ __managed__ int *devIpiv, devInfo;

int main(){
  hipsolverHandle_t handle;
  hipsolverStatus_t status;
  status = hipsolverDnCreate(&handle);
  
  int Lwork;

  status = hipsolverDnDgetrf_bufferSize(handle, 4, 4, A2, 4, &Lwork );

  hipMallocManaged(&Workspace, (size_t) Lwork*sizeof(double));
  hipMallocManaged(&devIpiv, (size_t) 4*sizeof(int));
  for(int i=0;i<4;i++){
    for(int j=0;j<4;j++){
      std::cout<<A2[j+4*i]<<"\t";
    }
    std::cout<<std::endl;
  }
  std::cout<<std::endl;
  status = hipsolverDnDgetrf(handle, 4, 4, A2, 4, Workspace, devIpiv, &devInfo);
  for(int i=0;i<4;i++){
    for(int j=0;j<4;j++){
      std::cout<<A2[i+4*j]<<"\t";
    }
    std::cout<<std::endl;
  }
  // A=LU => A=U
  
//  status = cusolverDnDpotrs(handle, CUBLAS_FILL_MODE_UPPER, 4, 1, A, 4, b, 4, &devInfo);

  status = hipsolverDnDgetrs(handle, HIPBLAS_OP_N, 4, 1, A2, 4, devIpiv, b, 4, &devInfo );
  hipDeviceSynchronize();
  if(status == HIPSOLVER_STATUS_SUCCESS){
    std::cout<<"Yay!"<<std::endl;
    std::cout<<devInfo<<std::endl;
    for(int i=0;i<4;i++){
      std::cout<<x[i]<<"\t"<<b[i]<<std::endl;
    }
  }

 return 0;
}
