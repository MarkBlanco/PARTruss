#include "solveMatrix.h"
#include <hip/hip_runtime.h>
#include "hipsolver.h"
double *cu_A;
double *b;
double *Workspace;
int *devIpiv;
__device__ __managed__ int devInfo;

int die(hipsolverStatus_t status, int devInfo, hipsolverHandle_t handle);
int err(hipsolverStatus_t status, int devInfo);
void freeMem();

int solveMatrix(double *A_in, int n, double *b_in, double *x_out)
{
    hipsolverHandle_t handle;
    hipsolverStatus_t status;
    status = hipsolverDnCreate(&handle);
    if(status!=HIPSOLVER_STATUS_SUCCESS) return 0;
    int Lwork;

    hipMallocManaged(&cu_A, (size_t) (n*n*sizeof(double)));
    hipMallocManaged(&b, (size_t) (n*sizeof(double)));
    hipMallocManaged(&devIpiv, (size_t) n*sizeof(int));
    hipMemcpy(cu_A, A_in, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b, b_in, n*sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    status = hipsolverDnDgetrf_bufferSize(handle, n, n, cu_A, n, &Lwork );
    hipDeviceSynchronize();
    if(err(status,0))
        return die(status, 0, handle);
    hipMallocManaged(&Workspace, (size_t) Lwork*sizeof(double));
    status = hipsolverDnDgetrf(handle, n, n, cu_A, n, Workspace, devIpiv, &devInfo);
    hipDeviceSynchronize();
    if(err(status,0))
        return die(status, devInfo, handle);
    status = hipsolverDnDgetrs(handle, HIPBLAS_OP_T, n, 1, cu_A, n, devIpiv, b, n, &devInfo );
    hipDeviceSynchronize();
    if(err(status,0))
        return die(status, devInfo, handle);
    hipMemcpy(x_out, b, n*sizeof(double), hipMemcpyDeviceToHost);
    hipsolverDnDestroy(handle);
    freeMem();
    return 0;
}

int die(hipsolverStatus_t status, int devInfo, hipsolverHandle_t handle)
{
    hipsolverDnDestroy(handle);
    freeMem();
    return err(status, devInfo);
}

int err(hipsolverStatus_t status, int devInfo)
{
    if(status!=HIPSOLVER_STATUS_SUCCESS)
        return 1;
    if(devInfo!=0)
        return devInfo;
    return 0;
}
void freeMem()
{
    hipFree(cu_A);
    hipFree(b);
    hipFree(devIpiv);
}
