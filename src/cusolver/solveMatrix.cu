#include <hip/hip_runtime.h>
#include "hipsolver.h"

__device__ double *A;
__device__ double *b;
__device__ double *Workspace;
__device__ int *devIpiv;
__device__ __managed__ int devInfo;

int solveMatrix(double *A_in, int n, double *b_in, double *x_out){
  hipsolverHandle_t handle;
  hipsolverStatus_t status;
  status = hipsolverDnCreate(&handle);
  if(status!=HIPSOLVER_STATUS_SUCCESS) return 1;
  int Lwork;

  hipMallocManaged(&A, (size_t) (n*n*sizeof(double)));
  hipMallocManaged(&b, (size_t) (n*sizeof(double)));
  hipMallocManaged(&devIpiv, (size_t) n*sizeof(int));
  hipMemcpy(A, A_in, n*n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(b, b_in, n*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  status = hipsolverDnDgetrf_bufferSize(handle, n, n, A, n, &Lwork );
  hipDeviceSynchronize();
  if(status!=HIPSOLVER_STATUS_SUCCESS) return 1;
  hipMallocManaged(&Workspace, (size_t) Lwork*sizeof(double));
  status = hipsolverDnDgetrf(handle, n, n, A, n, Workspace, devIpiv, &devInfo);
  hipDeviceSynchronize();
  if(status!=HIPSOLVER_STATUS_SUCCESS) return 1;
  status = hipsolverDnDgetrs(handle, HIPBLAS_OP_T, n, 1, A, n, devIpiv, b, n, &devInfo );
  hipDeviceSynchronize();
  if(status!=HIPSOLVER_STATUS_SUCCESS) return 1;
  hipMemcpy(x_out, b, n*sizeof(double), hipMemcpyDeviceToHost);
  return 0;
}
