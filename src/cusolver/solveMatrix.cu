#include <hip/hip_runtime.h>
#include "hipsolver.h"

__device__ double *A;
__device__ double *b;
__device__ double *Workspace;
__device__ int *devIpiv;
__device__ __managed__ int devInfo;

int solveMatrix(double *A_in, int n, double *b_in, double *x_out){
  hipsolverHandle_t handle;
  hipsolverStatus_t status;
  status = hipsolverDnCreate(&handle);
  if(status!=HIPSOLVER_STATUS_SUCCESS) return 0;
  int Lwork;

  hipMallocManaged(&A, (size_t) (n*n*sizeof(double)));
  hipMallocManaged(&b, (size_t) (n*sizeof(double)));
  hipMallocManaged(&devIpiv, (size_t) n*sizeof(int));
  hipMemcpy(A, A_in, n*n*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(b, b_in, n*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  status = hipsolverDnDgetrf_bufferSize(handle, n, n, A, n, &Lwork );
  hipDeviceSynchronize();
  if(status!=HIPSOLVER_STATUS_SUCCESS){
    return 1;
    hipsolverDnDestroy(handle);
  }
  hipMallocManaged(&Workspace, (size_t) Lwork*sizeof(double));
  status = hipsolverDnDgetrf(handle, n, n, A, n, Workspace, devIpiv, &devInfo);
  hipDeviceSynchronize();
  if(status!=HIPSOLVER_STATUS_SUCCESS){
    return 1;
    hipsolverDnDestroy(handle);
  }
  if(devInfo!=0){
    return devInfo;
    hipsolverDnDestroy(handle);
  }
  status = hipsolverDnDgetrs(handle, HIPBLAS_OP_T, n, 1, A, n, devIpiv, b, n, &devInfo );
  hipDeviceSynchronize();
  if(status!=HIPSOLVER_STATUS_SUCCESS){
   return 1;
   hipsolverDnDestroy(handle);
  }
  if(devInfo!=0){
    return devInfo;
    hipsolverDnDestroy(handle);
  }
  hipMemcpy(x_out, b, n*sizeof(double), hipMemcpyDeviceToHost);
  hipsolverDnDestroy(handle);
  return 0;
}
