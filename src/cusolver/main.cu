
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>

extern int solveMatrix(double *A_in, int n, double *b_in, double *x_out);

using namespace std;
int main(int argc, char *argv[]){
  ifstream mtx(argv[1]);
  ifstream vec(argv[2]);
  vector<double> A;
  vector<double> b;
  string line;
  int n=0;
  double token;
  while(getline(mtx,line)){
    stringstream input;
    input.str(line);
    int count = 0;
    while(input>>token){
      A.push_back(token);
      count++;
    }
    if(n==0) n=count;
  }
  while(vec>>token){
    b.push_back(token);
  }
  for(int i=0;i<n;i++){
    for(int j=0;j<n;j++){
      cout<<A[4*i+j]<<"\t";
    }
    cout<<endl;
  }
  cout<<endl;
  for(int i=0;i<n;i++){
    cout<<b[i]<<endl;
  }
  cout<<endl;
  
  solveMatrix(&A[0], n, &b[0], &b[0]);
  for(int i=0;i<n;i++){
    cout<<b[i]<<endl;
  }

  return 0;
}
